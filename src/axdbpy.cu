#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/axdbpy.hpp"

#include "complex.hpp"

template <typename T>
__global__ void axdbpy_kernel(size_t n, const T alpha, const T* x, size_t incx, T beta, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        y[incy * index] = (alpha * x[incx * index]) / (beta + y[incy * index]);
    }
}

template <typename T>
__global__ void axdbpy_kernel1(size_t n, const T alpha, const T* x, T beta, T* y) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        y[index] = (alpha * x[index]) / (beta + y[index]);
    }
}

template <typename T>
void axdbpy_kernel_run(size_t n, T alpha, const T* x, size_t incx, T beta, T* y, size_t incy) {
    static int blockSize;
    static int minGridSize;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axdbpy_kernel1<T>, 0, 0);
        blockSize = blockSize > 256 ? 256 : blockSize;
    }

    const int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1) {
        axdbpy_kernel1<T><<<gridSize, blockSize>>>(n, alpha, x, beta, y);
    } else {
        axdbpy_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, beta, y, incy);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

#ifdef EGBLAS_HAS_HAXDBPY

void egblas_haxdbpy(size_t n, fp16 alpha, const fp16* x, size_t incx, fp16 beta, fp16* y, size_t incy) {
    axdbpy_kernel_run(n, alpha, x, incx, beta, y, incy);
}

#endif

#ifdef EGBLAS_HAS_BAXDBPY

void egblas_baxdbpy(size_t n, bf16 alpha, const bf16* x, size_t incx, bf16 beta, bf16* y, size_t incy) {
    axdbpy_kernel_run(n, alpha, x, incx, beta, y, incy);
}

#endif

void egblas_saxdbpy(size_t n, float alpha, const float* x, size_t incx, float beta, float* y, size_t incy) {
    axdbpy_kernel_run(n, alpha, x, incx, beta, y, incy);
}

void egblas_daxdbpy(size_t n, double alpha, const double* x, size_t incx, double beta, double* y, size_t incy) {
    axdbpy_kernel_run(n, alpha, x, incx, beta, y, incy);
}

void egblas_caxdbpy(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex beta, hipComplex* y, size_t incy) {
    axdbpy_kernel_run(n, alpha, x, incx, beta, y, incy);
}

void egblas_zaxdbpy(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex beta, hipDoubleComplex* y, size_t incy) {
    axdbpy_kernel_run(n, alpha, x, incx, beta, y, incy);
}
