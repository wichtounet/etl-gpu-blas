#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/axpy_3.hpp"

#include "complex.hpp"

template <typename T>
__global__ void axpy_3_kernel(size_t n, T alpha, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        yy[incyy * index] = alpha * x[incx * index] + y[incy * index];
    }
}

template <typename T>
__global__ void axpy_3_kernel_flat(size_t n, T alpha, const T* x, const T* y, T* yy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        yy[index] = alpha * x[index] + y[index];
    }
}

template <typename T>
__global__ void axpy_3_kernel1(size_t n, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        yy[incyy * index] = x[incx * index] + y[incy * index];
    }
}

template <typename T>
__global__ void axpy_3_kernel1_flat(size_t n, const T* x, const T* y, T* yy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        yy[index] = x[index] + y[index];
    }
}

template <typename T>
__global__ void axpy_3_kernel0(size_t n, const T* y, size_t incx, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        yy[incyy * index] = y[incx * index];
    }
}

template <typename T>
__global__ void axpy_3_kernel0_flat(size_t n, const T * y, T* yy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        yy[index] = y[index];
    }
}

template <typename T>
void axpy_3_kernel_run(size_t n, T alpha, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_3_kernel<T>, 0, 0);
    }

    int gridSize = ((n / incyy) + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1 && incyy == 1) {
        axpy_3_kernel_flat<T><<<gridSize, blockSize>>>(n, alpha, x, y, yy);
    } else {
        axpy_3_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy, yy, incyy);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void axpy_3_kernel1_run(size_t n, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_3_kernel1<T>, 0, 0);
    }

    int gridSize = ((n / incyy) + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1 && incyy == 1) {
        axpy_3_kernel1_flat<T><<<gridSize, blockSize>>>(n, x, y, yy);
    } else {
        axpy_3_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy, yy, incyy);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void axpy_3_kernel0_run(size_t n, const T * y, size_t incy, T* yy, size_t incyy) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_3_kernel0<T>, 0, 0);
    }

    int gridSize = ((n / incyy) + blockSize - 1) / blockSize;

    if (incy == 1 && incyy == 1 ) {
        axpy_3_kernel0_flat<T><<<gridSize, blockSize>>>(n, y, yy);
    } else {
        axpy_3_kernel0<T><<<gridSize, blockSize>>>(n, y, incy, yy, incyy);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

#ifdef EGBLAS_HAS_HAXPY_3

void egblas_haxpy_3(size_t n, fp16 alpha, const fp16* x, size_t incx, const fp16* y, size_t incy, fp16* yy, size_t incyy) {
    if (__low2float(alpha) == 1.0f && __high2float(alpha) == 1.0f) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (__low2float(alpha) == 0.0f) {
        axpy_3_kernel0_run(n, y, incy, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

#endif

#ifdef EGBLAS_HAS_BAXPY_3

void egblas_baxpy_3(size_t n, bf16 alpha, const bf16* x, size_t incx, const bf16* y, size_t incy, bf16* yy, size_t incyy) {
    if (__low2float(alpha) == 1.0f && __high2float(alpha) == 1.0f) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (__low2float(alpha) == 0.0f) {
        axpy_3_kernel0_run(n, y, incy, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

#endif

void egblas_saxpy_3(size_t n, float alpha, const float* x, size_t incx, const float* y, size_t incy, float* yy, size_t incyy) {
    if (alpha == 1.0f) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0.0f) {
        axpy_3_kernel0_run(n, y, incy, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_daxpy_3(size_t n, double alpha, const double* x, size_t incx, const double* y, size_t incy, double* yy, size_t incyy) {
    if (alpha == 1.0) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0.0) {
        axpy_3_kernel0_run(n, y, incy, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_caxpy_3(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, const hipComplex* y, size_t incy, hipComplex* yy, size_t incyy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        axpy_3_kernel0_run(n, y, incy, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_zaxpy_3(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, const hipDoubleComplex* y, size_t incy, hipDoubleComplex* yy, size_t incyy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        axpy_3_kernel0_run(n, y, incy, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_iaxpy_3(size_t n, int32_t alpha, const int32_t* x, size_t incx, const int32_t* y, size_t incy, int32_t* yy, size_t incyy) {
    if (alpha == 1) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0) {
        axpy_3_kernel0_run(n, y, incy, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_laxpy_3(size_t n, int64_t alpha, const int64_t* x, size_t incx, const int64_t* y, size_t incy, int64_t* yy, size_t incyy) {
    if (alpha == 1) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0) {
        axpy_3_kernel0_run(n, y, incy, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}
