#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/apxdbpy_3.hpp"

#include "complex.hpp"

template <typename T>
__global__ void apxdbpy_3_kernel(size_t n, const T alpha, const T* x, size_t incx, T beta, const T* y, size_t incy, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        yy[incy * index] = (alpha + x[incx * index]) / (beta + y[incy * index]);
    }
}

template <typename T>
void apxdbpy_3_kernel_run(size_t n, T alpha, const T* x, size_t incx, T beta, const T* y, size_t incy, T* yy, size_t incyy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, apxdbpy_3_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    apxdbpy_3_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, beta, y, incy, yy, incyy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

#ifdef EGBLAS_HAS_HAPXDBPY_3

void egblas_hapxdbpy_3(size_t n, fp16 alpha, const fp16* x, size_t incx, fp16 beta, const fp16* y, size_t incy, fp16* yy, size_t incyy) {
    apxdbpy_3_kernel_run(n, alpha, x, incx, beta, y, incy, yy, incyy);
}

#endif

#ifdef EGBLAS_HAS_BAPXDBPY_3

void egblas_bapxdbpy_3(size_t n, bf16 alpha, const bf16* x, size_t incx, bf16 beta, const bf16* y, size_t incy, bf16* yy, size_t incyy) {
    apxdbpy_3_kernel_run(n, alpha, x, incx, beta, y, incy, yy, incyy);
}

#endif

void egblas_sapxdbpy_3(size_t n, float alpha, const float* x, size_t incx, float beta, const float* y, size_t incy, float* yy, size_t incyy) {
    apxdbpy_3_kernel_run(n, alpha, x, incx, beta, y, incy, yy, incyy);
}

void egblas_dapxdbpy_3(size_t n, double alpha, const double* x, size_t incx, double beta, const double* y, size_t incy, double* yy, size_t incyy) {
    apxdbpy_3_kernel_run(n, alpha, x, incx, beta, y, incy, yy, incyy);
}

void egblas_capxdbpy_3(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex beta, const hipComplex* y, size_t incy, hipComplex* yy, size_t incyy) {
    apxdbpy_3_kernel_run(n, alpha, x, incx, beta, y, incy, yy, incyy);
}

void egblas_zapxdbpy_3(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex beta, const hipDoubleComplex* y, size_t incy, hipDoubleComplex* yy, size_t incyy) {
    apxdbpy_3_kernel_run(n, alpha, x, incx, beta, y, incy, yy, incyy);
}
