#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include "egblas/transpose_front.hpp"
#include "egblas/assert.hpp"
#include "egblas/utils.hpp"
#include "egblas/sum.hpp"
#include "egblas/cuda_check.hpp"

template <typename T>
__global__ void transpose_front_kernel(size_t M, size_t N, size_t K, const T* x, T* y) {
    const auto mk  = threadIdx.x + blockIdx.x * blockDim.x;

    if (mk < M * K) {
        // Note: Ideally, we would use 2D Indexing. But I can't get it to be as fast as the 1D index
        const size_t m = mk / K;
        const size_t k = mk % K;

        for (size_t n = 0; n < N; ++n) {
            // y(n, m) = x(m, n)
            // x[M, N, K]
            // y[N, M, K]

            y[n * (M * K) + m * K + k] = x[m * (N * K) + n * K + k];
        }
    }
}

void egblas_stranspose_front(size_t m, size_t n, size_t k, float* x, float* y) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &transpose_front_kernel<float>, 0, 0);

    int gridSize = (m * k + blockSize - 1) / blockSize;

    transpose_front_kernel<<<gridSize, blockSize>>>(m, n, k, x, y);
}

void egblas_dtranspose_front(size_t m, size_t n, size_t k, double* x, double* y) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &transpose_front_kernel<double>, 0, 0);

    int gridSize = (m * k + blockSize - 1) / blockSize;

    transpose_front_kernel<<<gridSize, blockSize>>>(m, n, k, x, y);
}
