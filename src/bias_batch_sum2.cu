#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2020 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include "egblas/assert.hpp"
#include "egblas/utils.hpp"
#include "egblas/sum.hpp"
#include "egblas/cuda_check.hpp"

#include "sum_reduce.hpp"

template <bool Mean, typename T>
__global__ void bias_batch_sum_kernel(size_t B, size_t N, const T* x, size_t incx, T* y, size_t incy) {
    auto n  = threadIdx.x + blockIdx.x * blockDim.x;

    if (n < N) {
        T sum = 0;

        for (size_t b = 0; b < B; ++b) {
            sum += x[(b * N + n) * incx];
        }

        if (Mean) {
            y[incy * n] = sum / B;
        } else {
            y[incy * n] = sum;
        }
    }
}

template <bool Mean, typename T>
__global__ void bias_batch_sum_kernel_flat(size_t B, size_t N, const T* x, T* y) {
    auto n  = threadIdx.x + blockIdx.x * blockDim.x;

    if (n < N) {
        T sum = 0;

        size_t b = 0;

        for (; b + 7 < B; b += 8) {
            sum += x[(b + 0) * N + n];
            sum += x[(b + 1) * N + n];
            sum += x[(b + 2) * N + n];
            sum += x[(b + 3) * N + n];
            sum += x[(b + 4) * N + n];
            sum += x[(b + 5) * N + n];
            sum += x[(b + 6) * N + n];
            sum += x[(b + 7) * N + n];
        }

        for (; b + 3 < B; b += 4) {
            sum += x[(b + 0) * N + n];
            sum += x[(b + 1) * N + n];
            sum += x[(b + 2) * N + n];
            sum += x[(b + 3) * N + n];
        }

        for (; b + 1 < B; b += 2) {
            sum += x[(b + 0) * N + n];
            sum += x[(b + 1) * N + n];
        }

        // Note: This should be a if, but using a if makes it slower
        for (; b < B; ++b) {
            sum += x[b * N + n];
        }

        if (Mean) {
            y[n] = sum / B;
        } else {
            y[n] = sum;
        }
    }
}

template <typename T>
__global__ void bias_batch_var_kernel(size_t M, size_t N, const T* a, size_t inca, const T* b, size_t incb, T* y, size_t incy) {
    auto n  = threadIdx.x + blockIdx.x * blockDim.x;

    if (n < N) {
        T sum = 0;

        for (size_t m = 0; m < M; ++m) {
            sum += (a[(m * N + n) * inca] - b[n * incb]) * (a[(m * N + n) * inca] - b[n * incb]);
        }

        y[incy * n] = sum / M;
    }
}

template <typename T>
__global__ void bias_batch_var_kernel_flat(size_t M, size_t N, const T* a, const T* b, T* y) {
    auto n  = threadIdx.x + blockIdx.x * blockDim.x;

    if (n < N) {
        T sum = 0;

        for (size_t m = 0; m < M; ++m) {
            sum += (a[m * N + n] - b[n]) * (a[m * N + n] - b[n]);
        }

        y[n] = sum / M;
    }
}

void egblas_sbias_batch_sum(size_t b, size_t n, float* x, size_t incx, float* y, size_t incy) {
    const int blockSize = 64;
    const int gridSize = (n + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1) {
        bias_batch_sum_kernel_flat<false><<<gridSize, blockSize>>>(b, n, x, y);
    } else {
        bias_batch_sum_kernel<false><<<gridSize, blockSize>>>(b, n, x, incx, y, incy);
    }
}

void egblas_dbias_batch_sum(size_t b, size_t n, double* x, size_t incx, double* y, size_t incy) {
    const int blockSize = 64;
    const int gridSize = (n + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1) {
        bias_batch_sum_kernel_flat<false><<<gridSize, blockSize>>>(b, n, x, y);
    } else {
        bias_batch_sum_kernel<false><<<gridSize, blockSize>>>(b, n, x, incx, y, incy);
    }
}

void egblas_sbias_batch_mean(size_t b, size_t n, float* x, size_t incx, float* y, size_t incy) {
    const int blockSize = 64;
    const int gridSize = (n + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1) {
        bias_batch_sum_kernel_flat<true><<<gridSize, blockSize>>>(b, n, x, y);
    } else {
        bias_batch_sum_kernel<true><<<gridSize, blockSize>>>(b, n, x, incx, y, incy);
    }
}

void egblas_dbias_batch_mean(size_t b, size_t n, double* x, size_t incx, double* y, size_t incy) {
    const int blockSize = 64;
    const int gridSize = (n + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1) {
        bias_batch_sum_kernel_flat<true><<<gridSize, blockSize>>>(b, n, x, y);
    } else {
        bias_batch_sum_kernel<true><<<gridSize, blockSize>>>(b, n, x, incx, y, incy);
    }
}

void egblas_sbias_batch_var(size_t m, size_t n, float* a, size_t inca, float* b, size_t incb, float* y, size_t incy) {
    const int blockSize = 64;
    const int gridSize = (n + blockSize - 1) / blockSize;

    if (inca == 1 && incb == 1 && incy == 1) {
        bias_batch_var_kernel_flat<<<gridSize, blockSize>>>(m, n, a, b, y);
    } else {
        bias_batch_var_kernel<<<gridSize, blockSize>>>(m, n, a, inca, b, incb, y, incy);
    }
}

void egblas_dbias_batch_var(size_t m, size_t n, double* a, size_t inca, double* b, size_t incb, double* y, size_t incy) {
    const int blockSize = 64;
    const int gridSize = (n + blockSize - 1) / blockSize;

    if (inca == 1 && incb == 1 && incy == 1) {
        bias_batch_var_kernel_flat<<<gridSize, blockSize>>>(m, n, a, b, y);
    } else {
        bias_batch_var_kernel<<<gridSize, blockSize>>>(m, n, a, inca, b, incb, y, incy);
    }
}
