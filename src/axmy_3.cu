#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/axmy_3.hpp"

#include "complex.hpp"

template <typename T>
__global__ void axmy_3_kernel(size_t n, T alpha, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        yy[incyy * index] = alpha * x[incx * index] * y[incy * index];
    }
}

template <typename T>
__global__ void axmy_3_kernel_flat(size_t n, T alpha, const T* x, const T* y, T* yy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        yy[index] = alpha * x[index] * y[index];
    }
}

template <typename T>
__global__ void axmy_3_kernel1(size_t n, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        yy[incyy * index] = x[incx * index] * y[incy * index];
    }
}

template <typename T>
__global__ void axmy_3_kernel1_flat(size_t n, const T* x, const T* y, T* yy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        yy[index] = x[index] * y[index];
    }
}

template <typename T>
__global__ void axmy_3_kernel0(size_t n, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        yy[incyy * index] = zero<T>();
    }
}

template <typename T>
__global__ void axmy_3_kernel0_flat(size_t n, T* yy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        yy[index] = zero<T>();
    }
}

template <typename T>
void axmy_3_kernel_run(size_t n, T alpha, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axmy_3_kernel<T>, 0, 0);
    }

    int gridSize = (n + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1 && incyy == 1) {
        axmy_3_kernel_flat<T><<<gridSize, blockSize>>>(n, alpha, x, y, yy);
    } else {
        axmy_3_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy, yy, incyy);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void axmy_3_kernel1_run(size_t n, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axmy_3_kernel1<T>, 0, 0);
        blockSize = blockSize < 256 ? blockSize : 256;
    }

    int gridSize = (n + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1 && incyy == 1) {
        axmy_3_kernel1_flat<T><<<gridSize, blockSize>>>(n, x, y, yy);
    } else {
        axmy_3_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy, yy, incyy);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void axmy_3_kernel0_run(size_t n, T* yy, size_t incyy) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axmy_3_kernel0<T>, 0, 0);
    }

    int gridSize = (n + blockSize - 1) / blockSize;

    if (incyy == 1) {
        axmy_3_kernel0_flat<T><<<gridSize, blockSize>>>(n, yy);
    } else {
        axmy_3_kernel0<T><<<gridSize, blockSize>>>(n, yy, incyy);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

#ifdef EGBLAS_HAS_HAXMY_3

void egblas_haxmy_3(size_t n, fp16 alpha, const fp16* x, size_t incx, const fp16* y, size_t incy, fp16* yy, size_t incyy) {
    if (__low2float(alpha) == 1.0f && __high2float(alpha) == 1.0f) {
        axmy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (__low2float(alpha) == 0.0f) {
        axmy_3_kernel0_run(n, yy, incyy);
    } else {
        axmy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

#endif

#ifdef EGBLAS_HAS_BAXMY_3

void egblas_baxmy_3(size_t n, bf16 alpha, const bf16* x, size_t incx, const bf16* y, size_t incy, bf16* yy, size_t incyy) {
    if (__low2float(alpha) == 1.0f && __high2float(alpha) == 1.0f) {
        axmy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (__low2float(alpha) == 0.0f) {
        axmy_3_kernel0_run(n, yy, incyy);
    } else {
        axmy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

#endif

void egblas_saxmy_3(size_t n, float alpha, const float* x, size_t incx, const float* y, size_t incy, float* yy, size_t incyy) {
    if (alpha == 1.0f) {
        axmy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0.0f) {
        axmy_3_kernel0_run(n, yy, incyy);
    } else {
        axmy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_daxmy_3(size_t n, double alpha, const double* x, size_t incx, const double* y, size_t incy, double* yy, size_t incyy) {
    if (alpha == 1.0) {
        axmy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0.0) {
        axmy_3_kernel0_run(n, yy, incyy);
    } else {
        axmy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_caxmy_3(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, const hipComplex* y, size_t incy, hipComplex* yy, size_t incyy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        axmy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        axmy_3_kernel0_run(n, yy, incyy);
    } else {
        axmy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_zaxmy_3(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, const hipDoubleComplex* y, size_t incy, hipDoubleComplex* yy, size_t incyy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        axmy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        axmy_3_kernel0_run(n, yy, incyy);
    } else {
        axmy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_iaxmy_3(size_t n, int32_t alpha, const int32_t* x, size_t incx, const int32_t* y, size_t incy, int32_t* yy, size_t incyy) {
    if (alpha == 1) {
        axmy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0) {
        axmy_3_kernel0_run(n, yy, incyy);
    } else {
        axmy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_laxmy_3(size_t n, int64_t alpha, const int64_t* x, size_t incx, const int64_t* y, size_t incy, int64_t* yy, size_t incyy) {
    if (alpha == 1) {
        axmy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0) {
        axmy_3_kernel0_run(n, yy, incyy);
    } else {
        axmy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}
