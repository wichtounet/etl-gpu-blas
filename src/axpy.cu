#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================
#include "egblas/axpy.hpp"
#include <iostream>

#include "complex.hpp"

static constexpr int MAX_BLOCK_SIZE = 256;

template <typename T>
__global__ void axpy_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        y[incy * index] = y[incy * index] + alpha * x[incx * index];
    }
}

template <typename T>
__global__ void axpy_kernel_flat(size_t n, T alpha, const T* x, T* y) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        y[index] = y[index] + alpha * x[index];
    }
}

template <typename T>
__global__ void axpy_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        y[incy * index] = x[incx * index] + y[incy * index];
    }
}

template <typename T>
void axpy_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_kernel<T>, 0, 0);
        blockSize = blockSize > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : blockSize;
    }

    const int gridSize = (n + blockSize - 1) / blockSize;

    axpy_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void axpy_kernel_run_flat(size_t n, T alpha, const T* x, T* y) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_kernel_flat<T>, 0, 0);
        blockSize = blockSize > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : blockSize;
    }

    const int gridSize = (n + blockSize - 1) / blockSize;

    axpy_kernel_flat<T><<<gridSize, blockSize>>>(n, alpha, x, y);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void axpy_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_kernel1<T>, 0, 0);
        blockSize = blockSize > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : blockSize;
    }

    const int gridSize = (n + blockSize - 1) / blockSize;
    axpy_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

#ifdef EGBLAS_HAS_HAXPY

void egblas_haxpy(size_t n, fp16 alpha, const fp16* x, size_t incx, fp16* y, size_t incy) {
    if (__low2float(alpha) == 1.0f && __high2float(alpha) == 1.0f) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (__low2float(alpha) == 0.0f && __high2float(alpha) == 0.0f) {
        return;
    } else if (incx == 1 && incy == 1) {
        axpy_kernel_run_flat(n, alpha, x, y);
    } else {
        axpy_kernel_run(n, alpha, x, incx, y, incy);
    }
}

#endif

#ifdef EGBLAS_HAS_BAXPY

void egblas_baxpy(size_t n, bf16 alpha, const bf16* x, size_t incx, bf16* y, size_t incy) {
    if (__low2float(alpha) == 1.0f && __high2float(alpha) == 1.0f) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (__low2float(alpha) == 0.0f && __high2float(alpha) == 0.0f) {
        return;
    } else if (incx == 1 && incy == 1) {
        axpy_kernel_run_flat(n, alpha, x, y);
    } else {
        axpy_kernel_run(n, alpha, x, incx, y, incy);
    }
}

#endif

void egblas_saxpy(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        return;
    } else {
        if (incx == 1 && incy == 1) {
            axpy_kernel_run_flat(n, alpha, x, y);
        } else {
            axpy_kernel_run(n, alpha, x, incx, y, incy);
        }
    }
}

void egblas_daxpy(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        return;
    } else {
        if (incx == 1 && incy == 1) {
            axpy_kernel_run_flat(n, alpha, x, y);
        } else {
            axpy_kernel_run(n, alpha, x, incx, y, incy);
        }
    }
}

void egblas_caxpy(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        return;
    } else {
        if (incx == 1 && incy == 1) {
            axpy_kernel_run_flat(n, alpha, x, y);
        } else {
            axpy_kernel_run(n, alpha, x, incx, y, incy);
        }
    }
}

void egblas_zaxpy(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        return;
    } else {
        if (incx == 1 && incy == 1) {
            axpy_kernel_run_flat(n, alpha, x, y);
        } else {
            axpy_kernel_run(n, alpha, x, incx, y, incy);
        }
    }
}

void egblas_oaxpy(size_t n, int8_t alpha, const int8_t* x, size_t incx, int8_t* y, size_t incy) {
    if (alpha == 1) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0) {
        return;
    } else {
        if (incx == 1 && incy == 1) {
            axpy_kernel_run_flat(n, alpha, x, y);
        } else {
            axpy_kernel_run(n, alpha, x, incx, y, incy);
        }
    }
}

void egblas_waxpy(size_t n, int16_t alpha, const int16_t* x, size_t incx, int16_t* y, size_t incy) {
    if (alpha == 1) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0) {
        return;
    } else {
        if (incx == 1 && incy == 1) {
            axpy_kernel_run_flat(n, alpha, x, y);
        } else {
            axpy_kernel_run(n, alpha, x, incx, y, incy);
        }
    }
}

void egblas_iaxpy(size_t n, int32_t alpha, const int32_t* x, size_t incx, int32_t* y, size_t incy) {
    if (alpha == 1) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0) {
        return;
    } else {
        if (incx == 1 && incy == 1) {
            axpy_kernel_run_flat(n, alpha, x, y);
        } else {
            axpy_kernel_run(n, alpha, x, incx, y, incy);
        }
    }
}

void egblas_laxpy(size_t n, int64_t alpha, const int64_t* x, size_t incx, int64_t* y, size_t incy) {
    if (alpha == 1) {
        axpy_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0) {
        return;
    } else {
        if (incx == 1 && incy == 1) {
            axpy_kernel_run_flat(n, alpha, x, y);
        } else {
            axpy_kernel_run(n, alpha, x, incx, y, incy);
        }
    }
}
